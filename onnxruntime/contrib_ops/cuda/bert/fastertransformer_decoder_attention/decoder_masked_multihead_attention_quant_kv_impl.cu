#include "hip/hip_runtime.h"
/*
 * The implementation of this file is based on code provided by https://github.com/NVIDIA/FasterTransformer
 *
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Modifications Copyright (c) Microsoft.
// Licensed under the MIT License.

// Modifications:
// (1) Removed some code paths from the original implementation that had features which is not supported by
//  corresponding ORT kernel - for example- CrossAttention support, FP8, INT8, supports, etc.
// (2) When dealing with masked tokens, this kernel implementation deviates from FasterTransformer by applying
// mask filter values. Appropriate commentary exists in the code below.

#include "contrib_ops/cuda/bert/rotary_embedding_util.h"
#include "decoder_masked_multihead_attention_impl.h"
#include "decoder_masked_multihead_attention_impl_utils.h"
#include <cfloat>

namespace onnxruntime {
namespace contrib {
namespace cuda {

using namespace decoder_masked_self_attention_details;

template <typename T>
struct TFloatTypeFrom {
};

template <>
struct TFloatTypeFrom<float> {
  using Type = float;
};

template <>
struct TFloatTypeFrom<uint16_t> {
  using Type = half;
};

inline __device__ __half2 DequantizeChar2(const char2 ch2, const float unit_scale) {
  return __float22half2_rn(float2{unit_scale * ch2.x, unit_scale * ch2.y});
}

template <typename TVec>
class QuantVec {};

struct __align__(4) Char2x2 {
  char2 x;
  char2 y;
};

struct __align__(8) Char2x4 {
  char2 x;
  char2 y;
  char2 z;
  char2 w;
};

template <>
class QuantVec<uint32_t> {
 public:
  using Type = char2;
};

template <>
class QuantVec<uint2> {
 public:
  using Type = Char2x2;
};

template <>
class QuantVec<uint4> {
 public:
  using Type = Char2x4;
};

template <typename TVec>
inline __device__ TVec DequantizeVec(const typename QuantVec<TVec>::Type quant_vec_m, const float unit_scale);

template <>
inline __device__ uint32_t DequantizeVec<uint32_t>(const char2 ch2, const float unit_scale) {
  union __align__(4) {
    __half2 h2;
    uint32_t whole;
  }
  uh;
  uh.h2 = DequantizeChar2(ch2, unit_scale);
  return uh.whole;
}

template <>
inline __device__ uint2 DequantizeVec<uint2>(const Char2x2 ch2x2, const float unit_scale) {
  union __align__(8) {
    struct __align__(8) {
      __half2 h2x;
      __half2 h2y;
    };
    uint2 whole;
  }
  vec;
  vec.h2x = DequantizeChar2(ch2x2.x, unit_scale);
  vec.h2y = DequantizeChar2(ch2x2.y, unit_scale);
  return vec.whole;
}

template <>
inline __device__ uint4 DequantizeVec<uint4>(const Char2x4 ch2x4, const float unit_scale) {
  union __align__(16) {
    struct __align__(16) {
      __half2 h2x;
      __half2 h2y;
      __half2 h2z;
      __half2 h2w;
    };
    uint4 whole;
  }
  vec;
  vec.h2x = DequantizeChar2(ch2x4.x, unit_scale);
  vec.h2y = DequantizeChar2(ch2x4.y, unit_scale);
  vec.h2z = DequantizeChar2(ch2x4.z, unit_scale);
  vec.h2w = DequantizeChar2(ch2x4.w, unit_scale);
  return vec.whole;
}

template <typename TVec>
inline __device__ TVec LoadQuantVec(const TVec* q8, const float unit_scale) {
  using TQuantVec = typename QuantVec<TVec>::Type;
  TQuantVec quant_vec = *(const TQuantVec*)q8;
  return DequantizeVec<TVec>(quant_vec, unit_scale);
}

template <typename TFp, typename TVec>
inline __device__ TFp MaxAbsFloat(const TVec v);

template <>
inline __device__ __half MaxAbsFloat(const uint32_t v) {
  union __align__(4) {
    __half2 h2;
    uint32_t whole;
  }
  uvec = {.whole = v};
  const __half2 h2 = __habs2(uvec.h2);
  return __hmax(h2.x, h2.y);
}

template <>
inline __device__ __half MaxAbsFloat(const uint2 v) {
  // make it simple rather than save one op
  return __hmax(MaxAbsFloat<__half, uint32_t>(v.x), MaxAbsFloat<__half, uint32_t>(v.y));
}

template <>
inline __device__ __half MaxAbsFloat(const uint4 v) {
  return __hmax(__hmax(MaxAbsFloat<__half, uint32_t>(v.x), MaxAbsFloat<__half, uint32_t>(v.y)),
                __hmax(MaxAbsFloat<__half, uint32_t>(v.z), MaxAbsFloat<__half, uint32_t>(v.w)));
}

template <typename TVec>
inline __device__ typename QuantVec<TVec>::Type Quantize(const TVec v, const float scale);

template <>
inline __device__ char2 Quantize(const uint32_t v, const float inv_unit_scale) {
  union __align__(4) {
    uint32_t u;
    __half2 h2;
  }
  uh2 = {v};
  float2 f2 = __half22float2(uh2.h2);
  return char2{(char)min(max(-127, __float2int_rn(inv_unit_scale * f2.x)), 127),
               (char)min(max(-127, __float2int_rn(inv_unit_scale * f2.y)), 127)};
}

template <>
inline __device__ Char2x2 Quantize(const uint2 v, const float inv_unit_scale) {
  Char2x2 ch2x2;
  ch2x2.x = Quantize<uint32_t>(v.x, inv_unit_scale);
  ch2x2.y = Quantize<uint32_t>(v.y, inv_unit_scale);
  return ch2x2;
}

template <>
inline __device__ Char2x4 Quantize(const uint4 v, const float inv_unit_scale) {
  Char2x4 ch2x4;
  ch2x4.x = Quantize<uint32_t>(v.x, inv_unit_scale);
  ch2x4.y = Quantize<uint32_t>(v.y, inv_unit_scale);
  ch2x4.z = Quantize<uint32_t>(v.z, inv_unit_scale);
  ch2x4.w = Quantize<uint32_t>(v.w, inv_unit_scale);
  return ch2x4;
}

template <typename TVec>
inline __device__ void QuantizeTo(int8_t* dst, const TVec v, const float inv_unit_scale) {
  using TQuantVec = typename QuantVec<TVec>::Type;
  TQuantVec quant_vec = Quantize(v, inv_unit_scale);
  *(TQuantVec*)dst = quant_vec;
}

template <
    // The type of the inputs. Supported types: half(uint16_t).
    typename T,
    // The hidden dimension per head.
    int head_size,
    // The number of threads per key.
    int THREADS_PER_KEY,
    // The number of threads per value.
    int THREADS_PER_VALUE,
    // The number of threads in a threadblock.
    int THREADS_PER_BLOCK,
    // The type of the scale in memory
    typename TScale>
__global__ void masked_multihead_attention_quant_kv_kernel(DecoderMaskedMultiHeadAttentionQuantKVParams params) {
  // This kernel contains some code that cannot be compiled on CUDA ARCH 5.3 or lower
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 530
  (void)(params);
#else
  using TQ8 = int8_t;  // quantized value type for K V cache
  using TFp = typename TFloatTypeFrom<T>::Type;

  // Make sure the hidden dimension per head is a multiple of the number of threads per key.
  static_assert(head_size % THREADS_PER_KEY == 0, "");

  // Make sure the hidden dimension per head is a multiple of the number of threads per value.
  static_assert(head_size % THREADS_PER_VALUE == 0, "");

  // The size of a warp.
  constexpr int WARP_SIZE = 32;

  // The number of warps in a threadblock.
  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  extern __shared__ char smem_[];

  // The shared memory for the Q*K^T values and partial logits in softmax.
  float* qk_smem = reinterpret_cast<float*>(smem_);

  // The shared memory for the logits. For FP32, that's the same buffer as qk_smem.
  char* logits_smem_ = smem_;

  if (sizeof(T) != 4) {
    // For fp16, we have allocated separate memory for logits - use it
    logits_smem_ += (((params.total_sequence_length + 3) / 4) * 16);
  }

  T* logits_smem = reinterpret_cast<T*>(logits_smem_);

  // The shared memory to do the final reduction for the output values. Reuse qk_smem.
  T* out_smem = reinterpret_cast<T*>(smem_);

  // The shared memory buffers for the block-wide reductions. One for max, one for sum.
  __shared__ float red_smem[WARPS_PER_BLOCK * 2];

  // A vector of Q or K elements for the current timestep.
  using Qk_vec_k = typename Qk_vec_k_<T, head_size>::Type;  // with kernel-used precision
  using Qk_vec_m = typename Qk_vec_m_<T, head_size>::Type;  // with memory-used precision

  // Use alignment for safely casting the shared buffers as Qk_vec_k.
  // Shared memory to store Q inputs.
  __shared__ __align__(sizeof(Qk_vec_k)) T q_smem[head_size];

  // The number of elements per vector.
  constexpr int QK_VEC_SIZE = sizeof(Qk_vec_m) / sizeof(T);
  // caller need to check that
  //    * params.quant_kv_block_size is power of 2 and > 0
  //    * params.quant_kv_block_size % QK_VEC_SIZE == 0
  //    * params.quant_kv_block_size % K_VEC_SIZE == 0
  //    * params.quant_kv_block_size % V_VEC_SIZE == 0
  //    * head_size % params.quant_kv_block_size == 0
  const int scales_per_head = head_size / params.quant_kv_block_size;

  // Make sure the hidden size per head is a multiple of the vector size.
  static_assert(head_size % QK_VEC_SIZE == 0, "");

  constexpr int QK_THREAD_COUNT = head_size / QK_VEC_SIZE;
  static_assert(QK_THREAD_COUNT <= THREADS_PER_BLOCK);

  // The layout of the cache is [B, H, head_size/x, L, x] with x == 4/8/16 for FP32/FP16/FP8. Since each thread
  // owns x elements, we have to decompose the linear index into chunks of x values and the posi-
  // tion of the thread in that chunk.

  static_assert(sizeof(T) <= 16);
  static_assert(sizeof(Qk_vec_m) <= 16);

  // The number of elements in a chunk of 16B (that's the x in the above formula).
  constexpr int QK_ELTS_IN_16B = 16 / sizeof(T);

  // The number of K vectors in 16B.
  constexpr int QK_VECS_IN_16B = 16 / sizeof(Qk_vec_m);

  // The batch/beam idx
  const int bi = blockIdx.y;

  // The beam idx
  // const int beami = bi % params.beam_width;

  // The "beam-aware" batch idx
  const int bbi = bi / params.beam_width;

  // The head.
  const int hi = blockIdx.x;

  // Combine the batch and the head indices.
  const int bhi = bi * params.num_heads + hi;

  // Combine the "beam-aware" batch idx and the head indices.
  const int bbhi = bbi * params.beam_width * params.num_heads + hi;

  const int input_beam_index = bi % params.beam_width;

  // The thread in the block.
  const int tidx = threadIdx.x;

  // While doing the product Q*K^T for the different keys we track the max.
  float qk_max = -FLT_MAX;

  float qk = 0.0F;

  int qkv_base_offset = params.is_mha && !params.is_packed_qkv
                            ? bi * params.hidden_size + hi * head_size
                            : bi * (3 * params.hidden_size) + hi * head_size;

  const size_t bi_total_seq_length = bi * params.total_sequence_length;

  const size_t bi_max_seq_length = bi * params.max_sequence_length;

  int tlength = params.is_cross_attention ? params.kv_sequence_length : params.past_sequence_length;

  // First QK_THREAD_COUNT load Q and K + the bias values for the current timestep.
  const bool is_active_qk_thread = tidx < QK_THREAD_COUNT;

  // The offset in the Q and K buffer also accounts for the batch.
  int qk_offset = qkv_base_offset + tidx * QK_VEC_SIZE;

  // Trigger the loads from the Q and K buffers.
  Qk_vec_k q;
  zero(q);

  if (is_active_qk_thread) {
    q = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.q)[qk_offset]));
  }

  // The offset in the bias buffer.
  int qk_bias_offset = hi * head_size + tidx * QK_VEC_SIZE;

  // Trigger the loads from the Q and K bias buffers.
  if (params.q_bias && is_active_qk_thread) {
    Qk_vec_k q_bias;

    q_bias = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.q_bias)[qk_bias_offset]));

    q = add_vec(q, q_bias);
  }

  TQ8* params_k_cache = reinterpret_cast<TQ8*>(params.k_cache);

  const float inv_sqrt_dh = params.scale;

  if (is_active_qk_thread) {
    // Store the Q values to shared memory.
    *reinterpret_cast<Qk_vec_k*>(&q_smem[tidx * QK_VEC_SIZE]) = q;
  }

  if (!params.is_cross_attention) {
    Qk_vec_k k;

    if (is_active_qk_thread) {
      k = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.k)[qk_offset]));

      if (params.k_bias) {
        Qk_vec_k k_bias = vec_conversion<Qk_vec_k, Qk_vec_m>(
            *reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.k_bias)[qk_bias_offset]));
        k = add_vec(k, k_bias);
      }
    } else {
      zero(k);
    }

    if (params.rotary_embedding_dim > 0) {
      const bool do_rotary = is_active_qk_thread && QK_VEC_SIZE * tidx < params.rotary_embedding_dim;

      T* q_smem = reinterpret_cast<T*>(smem_);
      T* k_smem = q_smem + params.rotary_embedding_dim;

      const int half_rotary_dim = params.rotary_embedding_dim / 2;
      const int half_idx = (tidx * QK_VEC_SIZE) / half_rotary_dim;
      const int intra_half_idx = (tidx * QK_VEC_SIZE) % half_rotary_dim;
      const int smem_pitch = half_rotary_dim;

      assert(half_rotary_dim % QK_VEC_SIZE == 0);

      if (do_rotary) {
        *reinterpret_cast<Qk_vec_k*>(q_smem + half_idx * smem_pitch + intra_half_idx) = q;
        *reinterpret_cast<Qk_vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx) = k;
      }

      __syncthreads();

      const int transpose_idx = half_idx * (half_rotary_dim / 2) + intra_half_idx / 2;
      constexpr int tidx_factor = (QK_VEC_SIZE > 1) ? QK_VEC_SIZE / 2 : 1;

      if (do_rotary) {
        vec_from_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
        vec_from_smem_transpose(k, k_smem, transpose_idx, smem_pitch);

        apply_rotary_embedding(
            q, k, transpose_idx / tidx_factor, params.rotary_embedding_dim, params.t_step);

        write_smem_transpose(k, k_smem, transpose_idx, smem_pitch);
        write_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
      }

      __syncthreads();

      if (do_rotary) {
        q = *reinterpret_cast<Qk_vec_k*>(q_smem + half_idx * smem_pitch + intra_half_idx);
        k = *reinterpret_cast<Qk_vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx);
      }

      __syncthreads();
    }

    float max_abs_k = (float)MaxAbsFloat<TFp, Qk_vec_k>(k);
    // Perform the final reduction to compute the max inside each warp.
    const int qk_threads_per_scale = params.quant_kv_block_size / QK_VEC_SIZE;
    const int qk_threads_per_scale_in_warp = min(qk_threads_per_scale, WARP_SIZE);
    for (int mask = qk_threads_per_scale_in_warp / 2; mask >= 1; mask /= 2) {
      max_abs_k = fmaxf(max_abs_k, __shfl_xor_sync(uint32_t(-1), max_abs_k, mask));
    }

    if (qk_threads_per_scale > WARP_SIZE) {
      const int warp = tidx / WARP_SIZE;
      const int lane = tidx % WARP_SIZE;
      if (lane == 0) {
        red_smem[warp] = max_abs_k;
      }
      __syncthreads();

      // The warps finalize the reduction.
      max_abs_k = ((lane < WARPS_PER_BLOCK) ? red_smem[lane] : -FLT_MAX);

#pragma unroll
      for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
        max_abs_k = fmaxf(max_abs_k, __shfl_xor_sync(uint32_t(-1), max_abs_k, mask));
      }

      // Broadcast to all the threads in the warp.
      max_abs_k = __shfl_sync(uint32_t(-1), max_abs_k, 0);
    }

    if (is_active_qk_thread) {
      // Write the K values to the global memory cache.
      // NOTE: The stores are uncoalesced as we have multiple chunks of 16B spread across the memory
      // system. We designed it this way as it allows much better memory loads (and there are many
      // more loads) + the stores are really "write and forget" since we won't need the ack before
      // the end of the kernel. There's plenty of time for the transactions to complete.

      // The 16B chunk written by the thread.
      int co = tidx / QK_VECS_IN_16B;

      // The position of the thread in that 16B chunk.
      int ci = tidx % QK_VECS_IN_16B * QK_VEC_SIZE;

      // Two chunks are separated by L * x elements. A thread write QK_VEC_SIZE elements.
      int offset = bhi * params.max_sequence_length * head_size + co * params.max_sequence_length * QK_ELTS_IN_16B +
                   tlength * QK_ELTS_IN_16B + ci;
      // Trigger the stores to global memory.
      const float inv_unit_scale_k = (max_abs_k ? (127.0f / max_abs_k) : max_abs_k);
      QuantizeTo(&params_k_cache[offset], k, inv_unit_scale_k);
      if (tidx % qk_threads_per_scale == 0) {
        const int scale_offset = (bhi * params.max_sequence_length + tlength) * scales_per_head + tidx / qk_threads_per_scale;
        *(((TScale*)params.k_scale) + scale_offset) = (TScale)(max_abs_k / 127.0f);
      }

      // Compute \sum_i Q[i] * K^T[i] for the current timestep.
      using Qk_vec_acum = Qk_vec_k;
      qk = dot<Qk_vec_acum, Qk_vec_k>(q, k);

      if (QK_THREAD_COUNT <= WARP_SIZE) {
#pragma unroll
        for (int mask = QK_THREAD_COUNT / 2; mask >= 1; mask /= 2) {
          qk += __shfl_xor_sync(shfl_mask(QK_THREAD_COUNT), qk, mask);
        }
      }
    }

    if (QK_THREAD_COUNT > WARP_SIZE) {
      constexpr int WARPS_PER_RED = (QK_THREAD_COUNT + WARP_SIZE - 1) / WARP_SIZE;
      qk = block_sum<WARPS_PER_RED>(&red_smem[WARPS_PER_RED], qk);
    }

    // Store that value in shared memory. Keep the Q*K^T value in register for softmax.
    if (tidx == 0) {
      // Normalize qk.
      qk *= inv_sqrt_dh;
      if (params.relative_attention_bias != nullptr) {
        qk = add_vec(qk,
                     reinterpret_cast<T*>(params.relative_attention_bias)[hi * params.sequence_length * params.total_sequence_length + tlength]);
      }
      qk_max = qk;
      qk_smem[tlength] = qk;
    }
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The type of queries and keys for the math in the Q*K^T product.
  using K_vec_k = typename K_vec_k_<T, THREADS_PER_KEY>::Type;
  using K_vec_m = typename K_vec_m_<T, THREADS_PER_KEY>::Type;

  // The number of elements per vector.
  constexpr int K_VEC_SIZE = sizeof(K_vec_m) / sizeof(T);

  // Make sure the hidden size per head is a multiple of the vector size.
  static_assert(head_size % K_VEC_SIZE == 0, "");

  // The number of elements per thread.
  constexpr int K_ELTS_PER_THREAD = head_size / THREADS_PER_KEY;

  // The number of vectors per thread.
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;

  // The position the first key loaded by each thread from the cache buffer (for this B * H).
  int ko = tidx / THREADS_PER_KEY;

  // The position of the thread in the chunk of keys.
  int ki = tidx % THREADS_PER_KEY * K_VEC_SIZE;

  static_assert(head_size == THREADS_PER_KEY * K_VEC_SIZE * K_VECS_PER_THREAD);

  // Load the Q values from shared memory. The values are reused during the loop on K.
  K_vec_k q_vec[K_VECS_PER_THREAD];
#pragma unroll
  for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
    q_vec[ii] = *reinterpret_cast<const K_vec_k*>(&q_smem[ki + ii * THREADS_PER_KEY * K_VEC_SIZE]);
  }

  // The number of timesteps loaded per iteration.
  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;

  // The number of keys per warp.
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;

  // Base pointer for the beam's batch, before offsetting with indirection buffer
  TQ8* k_cache_batch = &params_k_cache[bbhi * params.max_sequence_length * head_size + ki];

  // Pick a number of keys to make sure all the threads of a warp enter (due to shfl_sync).
  int ti_end = ((tlength + K_PER_WARP - 1) / K_PER_WARP) * K_PER_WARP;

  // Iterate over the keys/timesteps to compute the various (Q*K^T)_{ti} values.
  bool has_beams = params.cache_indir != nullptr && !params.is_cross_attention;
  const int* beam_indices = has_beams ? &params.cache_indir[bi_max_seq_length] : nullptr;

  for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
    bool is_masked = (params.mask != nullptr) && (params.mask[bi_total_seq_length + ti] == 0) && (ti < tlength);
    const int mapped_beam_index = (has_beams && ti < tlength) ? beam_indices[ti] : input_beam_index;
    const int beam_offset = mapped_beam_index * params.num_heads * params.max_sequence_length * head_size;

    // The keys loaded from the key cache.
    K_vec_k k_vec[K_VECS_PER_THREAD];

    if (ti < tlength) {
      const int mapped_bhi = bbhi + mapped_beam_index * params.num_heads;
      const TScale* scales_in_head = ((const TScale*)params.k_scale) + ((mapped_bhi * params.max_sequence_length + ti) * scales_per_head);
      float unit_scale_k = 0.0f;
      int in_head_elem_idx = ki;
      int renew_scale_elem_idx = 0; // reload scale when in_head_elem_idx >= renew_scale_elem_idx
#pragma unroll
      for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
        if (in_head_elem_idx >= renew_scale_elem_idx) {
          int in_head_scale_idx = in_head_elem_idx / params.quant_kv_block_size;
          renew_scale_elem_idx = (in_head_scale_idx + 1) * params.quant_kv_block_size;
          unit_scale_k = (float)scales_in_head[in_head_scale_idx];
        }
        in_head_elem_idx += QK_ELTS_IN_16B;
        int jj = ii * params.max_sequence_length + ti;
        k_vec[ii] = LoadQuantVec((const K_vec_k*)(&k_cache_batch[beam_offset + jj * QK_ELTS_IN_16B]), unit_scale_k);
      }
    } else {
#pragma unroll
      for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
        zero(k_vec[ii]);
      }
    }

    // Perform the dot product and normalize qk.
    // WARNING: ALL THE THREADS OF A WARP MUST ENTER!!!
    float qk = Qk_dot<T, THREADS_PER_KEY>::dot(q_vec, k_vec) * inv_sqrt_dh;

    // This is a deviation from FasterTransformer kernel implementation
    // but this aligns with ORT's other Attention kernels which strives to
    // mimic PyTorch when dealing with mask filter values
    if (is_masked) {
      qk += params.mask_filter_value;
    }

    // Store the product to shared memory. There's one qk value per timestep. Update the max.
    if (ti < tlength && tidx % THREADS_PER_KEY == 0) {
      if (params.relative_attention_bias != nullptr) {
        qk = add_vec(qk,
                     reinterpret_cast<T*>(params.relative_attention_bias)[hi * params.sequence_length * params.total_sequence_length + ti]);
      }
      qk_max = fmaxf(qk_max, qk);
      qk_smem[ti] = qk;
    }
  }

  // Perform the final reduction to compute the max inside each warp.
  //
  // NOTE: In a group of THREADS_PER_KEY threads, the leader already has the max value for the
  // group so it's not needed to run the reduction inside the group (again).
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Decompose the thread index into warp and lane.
  const int warp = tidx / WARP_SIZE;
  const int lane = tidx % WARP_SIZE;

  // The warp leader writes the max to shared memory.
  if (lane == 0) {
    red_smem[warp] = qk_max;
  }

  // Make sure the products are in shared memory.
  __syncthreads();

  // The warps finalize the reduction.
  qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Broadcast to all the threads in the warp.
  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  // Compute the logits and start the sum.
  float sum = 0.f;
  int sum_tlength = params.is_cross_attention ? tlength - 1 : tlength;
  for (int ti = tidx; ti <= sum_tlength; ti += THREADS_PER_BLOCK) {
    // This is a deviation from FasterTransformer kernel implementation
    // but this aligns with ORT's other Attention kernels which strives to
    // mimic PyTorch when dealing with mask filter values
    float logit = __expf(qk_smem[ti] - qk_max);
    sum += logit;
    qk_smem[ti] = logit;
  }

  // Compute the sum.
  sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], sum);

  // Normalize the logits.
  float inv_sum = __fdividef(1.f, sum + 1.e-6f);
  for (int ti = tidx; ti <= sum_tlength; ti += THREADS_PER_BLOCK) {
    float logit = qk_smem[ti] * inv_sum;
    ConvertFromFloat(logits_smem[ti], logit);
  }

  // Put Values part below so we leverage __syncthreads
  // from the previous step

  // The number of elements per vector.
  constexpr int V_VEC_SIZE = head_size / THREADS_PER_VALUE;

  // A vector of V elements for the current timestep.
  using V_vec_k = typename V_vec_k_<T, V_VEC_SIZE>::Type;
  using V_vec_m = typename V_vec_m_<T, V_VEC_SIZE>::Type;

  // The value computed by this thread.
  int vo = tidx / THREADS_PER_VALUE;

  // The hidden dimensions computed by this particular thread.
  int vi = tidx % THREADS_PER_VALUE * V_VEC_SIZE;

  // The base pointer for the value in the cache buffer.
  TQ8* params_v_cache = reinterpret_cast<TQ8*>(params.v_cache);

  TQ8* v_cache = &params_v_cache[bhi * params.max_sequence_length * head_size + vi];

  // Base pointer for the beam's batch, before offsetting with indirection buffer
  TQ8* v_cache_batch = &params_v_cache[bbhi * params.max_sequence_length * head_size + vi];

  // The number of values processed per iteration of the loop.
  constexpr int V_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_VALUE;

  // One group of threads computes the product(s) for the current timestep.
  V_vec_k v_bias;
  if (params.v_bias && !params.is_cross_attention) {
    zero(v_bias);

    T* params_v_bias = reinterpret_cast<T*>(params.v_bias);

    if (vo == tlength % V_PER_ITER) {
      v_bias = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&params_v_bias[hi * head_size + vi]));
    }
  }

  // From previous, before values, step
  // Also make sure the logits are in shared memory.
  __syncthreads();

  using V_vec_acum = typename V_vec_acum_fp32_<V_vec_k>::Type;

  // The partial outputs computed by each thread.
  V_vec_acum out;
  zero(out);

  // Loop over the timesteps to compute the partial outputs.
  for (int ti = vo; ti < tlength; ti += V_PER_ITER) {
    // Fetch offset based on cache_indir when beam sampling
    const int mapped_beam_index = has_beams ? params.cache_indir[bi_max_seq_length + ti] : input_beam_index;
    const int beam_offset = mapped_beam_index * params.num_heads * params.max_sequence_length * head_size;

    const int mapped_bhi = bbhi + mapped_beam_index * params.num_heads;
    const int scale_offset = (mapped_bhi * params.max_sequence_length + ti) * scales_per_head + vi / params.quant_kv_block_size;
    const float unit_scale_v = (float)*(((TScale*)params.v_scale) + scale_offset);

    // Load the values from the cache.
    // V_vec_k v = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&v_cache_batch[beam_offset + ti * head_size]));
    V_vec_k v = LoadQuantVec((const V_vec_k*)(&v_cache_batch[beam_offset + ti * head_size]), unit_scale_v);

    // Load the logits from shared memory.
    T logit = logits_smem[ti];
    out = fma(logit, v, out);
  }

  // One group of threads computes the product(s) for the current timestep.
  if ((vo == tlength % V_PER_ITER) && !params.is_cross_attention) {
    const auto v_offset = qkv_base_offset + vi;

    V_vec_k v;
    v = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&reinterpret_cast<T*>(params.v)[v_offset]));
    if (params.v_bias) {
      v = add_vec(v, v_bias);
    }

    static_assert(THREADS_PER_VALUE <= WARP_SIZE);
    float max_abs_v = (float)MaxAbsFloat<TFp, V_vec_k>(v);
    const uint32_t group_id = (tidx % WARP_SIZE) / THREADS_PER_VALUE;
    const uint32_t group_masks = ((1u << THREADS_PER_VALUE) - 1) << (group_id * THREADS_PER_VALUE);
#pragma unroll
    for (int mask = THREADS_PER_VALUE / 2; mask >= 1; mask /= 2) {
      max_abs_v = fmaxf(max_abs_v, __shfl_xor_sync(group_masks, max_abs_v, mask, THREADS_PER_VALUE));
    }

    // Store the values with bias back to global memory in the cache for V.
    //*reinterpret_cast<V_vec_m*>(&v_cache[tlength * head_size]) = vec_conversion<V_vec_m, V_vec_k>(v);
    const float inv_unit_scale_v = (max_abs_v ? (127.0f / max_abs_v) : max_abs_v);
    QuantizeTo(&v_cache[tlength * head_size], v, inv_unit_scale_v);
    if (vi % params.quant_kv_block_size == 0) {
      const int scales_per_head = head_size / params.quant_kv_block_size;
      const int scale_offset = (bhi * params.max_sequence_length + tlength) * scales_per_head + vi / params.quant_kv_block_size;
      *(((TScale*)params.v_scale) + scale_offset) = (TScale)(max_abs_v / 127.0f);
    }

    // Initialize the output value with the current timestep.
    out = fma(logits_smem[tlength], v, out);
  }

  // Make sure we can start writing to shared memory.
  __syncthreads();

  // Run the final reduction amongst the different groups computing different partial outputs.
#pragma unroll
  for (int active_groups = V_PER_ITER; active_groups >= 2; active_groups /= 2) {
    // The midpoint in the number of active groups.
    int midpoint = active_groups / 2;

    // The upper part of active threads store to shared memory.
    if (vo >= midpoint && vo < active_groups) {
      ConvertFromFloat(*reinterpret_cast<V_vec_k*>(&out_smem[(vo - midpoint) * head_size + vi]), out);
    }
    __syncthreads();

    // The bottom warps update their values.
    if (vo < midpoint) {
      out = add_vec(*reinterpret_cast<const V_vec_k*>(&out_smem[vo * head_size + vi]), out);
    }
    __syncthreads();
  }

  // Output the final values.
  T* params_out = reinterpret_cast<T*>(params.out);
  if (vo == 0) {
    ConvertFromFloat(*reinterpret_cast<V_vec_m*>(&params_out[bhi * head_size + vi]), out);
  }
#endif
}

typedef __half TQuantKVScale;

// Template instantiation(s)

#define Instantiate(TQuantKVScale)                                                                                                                                    \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 32, 4, 4, 64, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);    \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 32, 2, 4, 128, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);   \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 32, 1, 4, 256, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);   \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 64, 4, 8, 64, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);    \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 64, 2, 8, 128, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);   \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 64, 1, 8, 256, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);   \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 128, 4, 16, 64, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);  \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 128, 2, 16, 128, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params); \
  template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 128, 1, 16, 256, TQuantKVScale>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

Instantiate(__half);
Instantiate(float);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
