#include "hip/hip_runtime.h"
/*
 * The implementation of this file is based on code provided by https://github.com/NVIDIA/FasterTransformer
 *
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// Modifications Copyright (c) Microsoft.
// Licensed under the MIT License.

// Modifications:
// (1) Removed some code paths from the original implementation that had features which is not supported by
//  corresponding ORT kernel - for example- CrossAttention support, FP8, INT8, supports, etc.
// (2) When dealing with masked tokens, this kernel implementation deviates from FasterTransformer by applying
// mask filter values. Appropriate commentary exists in the code below.

#include "contrib_ops/cuda/bert/rotary_embedding_util.h"
#include "decoder_masked_multihead_attention_impl.h"
#include "decoder_masked_multihead_attention_impl_utils.h"
#include <cfloat>

namespace onnxruntime {
namespace contrib {
namespace cuda {

using namespace decoder_masked_self_attention_details;

template <typename T>
struct TFloatTypeFrom{
};

template <>
struct TFloatTypeFrom<float> {
  using Type = float;
};

template <>
struct TFloatTypeFrom<uint16_t> {
  using Type = half;
};

inline __device__ __half2 DequantizeChar2(char2 ch2, const __half2 scale2) {
  // For speed test, for nh=12, hs=128, batch=1, prompt=1000, max_seq_len=1024

  // A100: avg inference time: 0.239 ms, (pure fp16: 0.161)
  return __h2div(__hmul2(__half2{ch2.x, ch2.y}, scale2), __half2{127, 127});

  // A100: avg inference time: 0.203,
  // return __half2{ch2.x / 64.0f, ch2.y / 64.0f};

  // below goes to 0.60 ms
  // return __half2{ch2.x / 127.0f, ch2.y / 127.0f};

  // below goes to 0.196 ms
  // return __half2{ch2.x, ch2.x};
}

template <typename TVec_mem>
class QuantVec {};

struct __align__(4) Char2x2 {
  char2 x;
  char2 y;
};

struct __align__(8) Char2x4 {
  char2 x;
  char2 y;
  char2 z;
  char2 w;
};

template <>
class QuantVec<uint32_t> {
public:
  using Type = char2;
};

template <>
class QuantVec<uint2> {
public:
  using Type = Char2x2;
};

template <>
class QuantVec<uint4> {
public:
  using Type = Char2x4;
};


template <typename TVec_mem>
inline __device__ TVec_mem DequantizeVec(const typename QuantVec<TVec_mem>::Type qv, const __half2 scale2);

template <>
inline __device__ uint32_t DequantizeVec<uint32_t>(const char2 ch2, const __half2 scale2) {
  union __align__(4) {
    __half2 h2;
    uint32_t whole;
  } uh;
  uh.h2 = DequantizeChar2(ch2, scale2);
  return uh.whole;
}

template <>
inline __device__ uint2 DequantizeVec<uint2>(const Char2x2 ch2x2, const __half2 scale2) {
  union __align__(8) {
    struct __align__(8) {
      __half2 h2x;
      __half2 h2y;
    };
    uint2 whole;
  } vec;
  vec.h2x = DequantizeChar2(ch2x2.x, scale2);
  vec.h2y = DequantizeChar2(ch2x2.y, scale2);
  return vec.whole;
}


template <>
inline __device__ uint4 DequantizeVec<uint4>(const Char2x4 ch2x4, const __half2 scale2) {
  union __align__(16) {
    struct __align__(16) {
      __half2 h2x;
      __half2 h2y;
      __half2 h2z;
      __half2 h2w;
    };
    uint4 whole;
  } vec;
  vec.h2x = DequantizeChar2(ch2x4.x, scale2);
  vec.h2y = DequantizeChar2(ch2x4.y, scale2);
  vec.h2z = DequantizeChar2(ch2x4.z, scale2);
  vec.h2w = DequantizeChar2(ch2x4.w, scale2);
  return vec.whole;
}


template <typename TVec_mem>
inline __device__ TVec_mem LoadQ8(const TVec_mem* q8, const __half2 scale2) {
  using Quant_Vec_mem = typename QuantVec<TVec_mem>::Type;
  Quant_Vec_mem qv = *(const Quant_Vec_mem*)q8;
  return DequantizeVec<TVec_mem>(qv, scale2);
}


// template <>
// inline __device__ uint32_t LoadQ8(const uint32_t* q8, const __half2 scale2) {
//   char2 ch2 = *(const char2*)q8;

//   union __align__(4) {
//     __half2 h2;
//     uint32_t whole;
//   } vec;
//   vec.h2 = DequantizeChar2(ch2, scale2);
//   return vec.whole;
// }

// template <>
// inline __device__ uint2 LoadQ8(const uint2* q8, const __half2 scale2) {
//   struct __align__(4) Char2x2 {
//     char2 x;
//     char2 y;
//   } ch2x2;
//   ch2x2 = *(const Char2x2 *)q8;

//   union __align__(8) {
//     struct __align__(8) {
//       __half2 h2x;
//       __half2 h2y;
//     };
//     uint2 whole;
//   } vec;
//   vec.h2x = DequantizeChar2(ch2x2.x, scale2);
//   vec.h2y = DequantizeChar2(ch2x2.y, scale2);
//   return vec.whole;
// }

// template <>
// inline __device__ uint4 LoadQ8(const uint4* q8, const __half2 scale2) {
//   struct __align__(8) Char2x4 {
//     char2 x;
//     char2 y;
//     char2 z;
//     char2 w;
//   } ch2x4;
//   ch2x4 = *(const Char2x4 *)q8;

//   union __align__(16) {
//     struct __align__(16) {
//       __half2 h2x;
//       __half2 h2y;
//       __half2 h2z;
//       __half2 h2w;
//     };
//     uint4 whole;
//   } vec;
//   vec.h2x = DequantizeChar2(ch2x4.x, scale2);
//   vec.h2y = DequantizeChar2(ch2x4.y, scale2);
//   vec.h2z = DequantizeChar2(ch2x4.z, scale2);
//   vec.h2w = DequantizeChar2(ch2x4.w, scale2);
//   return vec.whole;
// }

template <typename TVec>
inline __device__ __half MaxAbsFloat(const TVec v);

template <>
inline __device__ __half MaxAbsFloat(const uint32_t v) {
  union __align__(4) {
    __half2 h2;
    uint32_t whole;
  } uvec;
  uvec.whole = v;
  uvec.h2 = __habs2(uvec.h2);
  return __hmax(uvec.h2.x, uvec.h2.y);
}

template <>
inline __device__ __half MaxAbsFloat(const uint2 v) {
  union __align__(8) {
    struct __align__(8) {
      __half2 h2x;
      __half2 h2y;
    };
    uint2 whole;
  } uvec;
  uvec.whole = v;
  uvec.h2x = __habs2(uvec.h2x);
  uvec.h2y = __habs2(uvec.h2y);
  uvec.h2x = __hmax2(uvec.h2x, uvec.h2y);
  return __hmax(uvec.h2x.x, uvec.h2x.y);
}

template <>
inline __device__ __half MaxAbsFloat(const uint4 v) {
  union __align__(16) {
    struct __align__(16) {
      __half2 h2x;
      __half2 h2y;
      __half2 h2z;
      __half2 h2w;
    };
    uint4 whole;
  } uvec;
  uvec.whole = v;
  uvec.h2x = __habs2(uvec.h2x);
  uvec.h2y = __habs2(uvec.h2y);
  uvec.h2z = __habs2(uvec.h2z);
  uvec.h2w = __habs2(uvec.h2w);
  uvec.h2x = __hmax2(uvec.h2x, uvec.h2y);
  uvec.h2z = __hmax2(uvec.h2z, uvec.h2w);
  uvec.h2x = __hmax2(uvec.h2x, uvec.h2z);
  return __hmax(uvec.h2x.x, uvec.h2x.y);
}

// scale2.x == scale2.y, and not zero
inline __device__ char2 QuantizeHalf2(const uint32_t v, const __half2 scale2) {
  union __align__(4) {
    uint32_t u;
    __half2 h2;
  } uh2;

  const __half2 one_two_seven{127, 127};
  uh2.u = v;
  uh2.h2 = __hmul2(__h2div(uh2.h2, scale2), one_two_seven);
  return char2{__half2char_rz(uh2.h2.x), __half2char_rz(uh2.h2.y)};
}

template <typename TVec>
inline __device__ void QuantizeTo(int8_t* dst, const TVec v, const __half2 scale2);

template <>
inline __device__ void QuantizeTo(int8_t* dst, const uint32_t v, const __half2 scale2) {
  union __align__(2) {
    char2 ch2;
    uint16_t whole;
  } uvec;

  if (scale2.x) {
    uvec.ch2 = QuantizeHalf2(v, scale2);
  } else {
    uvec.whole = 0;
  }
  *(uint16_t*)dst = uvec.whole;
}

template <>
inline __device__ void QuantizeTo(int8_t* dst, const uint2 v, const __half2 scale2) {
  union __align__(4) Char2x2 {
    struct __align__(4) {
      char2 x;
      char2 y;
    };
    uint32_t whole;
  } ch2x2;

  if (scale2.x) {
    ch2x2.x = QuantizeHalf2(v.x, scale2);
    ch2x2.y = QuantizeHalf2(v.y, scale2);
  } else {
    ch2x2.whole = 0;
  }
  *(uint32_t *)dst = ch2x2.whole;
}

template <>
inline __device__ void QuantizeTo(int8_t* dst, const uint4 v, const __half2 scale2) {
  union __align__(8) Char2x4 {
    struct __align__(8) {
      char2 x;
      char2 y;
      char2 z;
      char2 w;
    };
    uint2 whole;
  } ch2x4;

  if (scale2.x) {
    ch2x4.x = QuantizeHalf2(v.x, scale2);
    ch2x4.y = QuantizeHalf2(v.y, scale2);
    ch2x4.z = QuantizeHalf2(v.z, scale2);
    ch2x4.w = QuantizeHalf2(v.w, scale2);
  } else {
    ch2x4.whole.x = ch2x4.whole.y = 0;
  }
  *(uint2 *)dst = ch2x4.whole;
}

template <
    // The type of the inputs. Supported types: float and half(uint16_t).
    typename T,
    // The hidden dimension per head.
    int head_size,
    // The number of threads per key.
    int THREADS_PER_KEY,
    // The number of threads per value.
    int THREADS_PER_VALUE,
    // The number of threads in a threadblock.
    int THREADS_PER_BLOCK>
__global__ void masked_multihead_attention_quant_kv_kernel(DecoderMaskedMultiHeadAttentionQuantKVParams params) {
  // This kernel contains some code that cannot be compiled on CUDA ARCH 5.3 or lower
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 530
  (void)(params);
#else
  using TQ8 = int8_t; // quantized value type for K V cache
  using TFp = typename TFloatTypeFrom<T>::Type;

  // Make sure the hidden dimension per head is a multiple of the number of threads per key.
  static_assert(head_size % THREADS_PER_KEY == 0, "");

  // Make sure the hidden dimension per head is a multiple of the number of threads per value.
  static_assert(head_size % THREADS_PER_VALUE == 0, "");

  // The size of a warp.
  constexpr int WARP_SIZE = 32;

  // The number of warps in a threadblock.
  constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / WARP_SIZE;

  extern __shared__ char smem_[];

  // The shared memory for the Q*K^T values and partial logits in softmax.
  float* qk_smem = reinterpret_cast<float*>(smem_);

  // The shared memory for the logits. For FP32, that's the same buffer as qk_smem.
  char* logits_smem_ = smem_;

  if (sizeof(T) != 4) {
    // For fp16, we have allocated separate memory for logits - use it
    logits_smem_ += (((params.total_sequence_length + 3) / 4) * 16);
  }

  T* logits_smem = reinterpret_cast<T*>(logits_smem_);

  // The shared memory to do the final reduction for the output values. Reuse qk_smem.
  T* out_smem = reinterpret_cast<T*>(smem_);

  // The shared memory buffers for the block-wide reductions. One for max, one for sum.
  __shared__ float red_smem[WARPS_PER_BLOCK * 2];

  // A vector of Q or K elements for the current timestep.
  using Qk_vec_k = typename Qk_vec_k_<T, head_size>::Type;  // with kernel-used precision
  using Qk_vec_m = typename Qk_vec_m_<T, head_size>::Type;  // with memory-used precision

  // Use alignment for safely casting the shared buffers as Qk_vec_k.
  // Shared memory to store Q inputs.
  __shared__ __align__(sizeof(Qk_vec_k)) T q_smem[head_size];

  // The number of elements per vector.
  constexpr int QK_VEC_SIZE = sizeof(Qk_vec_m) / sizeof(T);
  // caller need to check that
  //    * params.quant_kv_block_size is power of 2 and > 0
  //    * params.quant_kv_block_size % QK_VEC_SIZE == 0
  //    * params.quant_kv_block_size % K_VEC_SIZE == 0
  //    * params.quant_kv_block_size % V_VEC_SIZE == 0
  //    * head_size % params.quant_kv_block_size == 0
  const int scales_per_head = head_size / params.quant_kv_block_size;


  // Make sure the hidden size per head is a multiple of the vector size.
  static_assert(head_size % QK_VEC_SIZE == 0, "");

  constexpr int QK_THREAD_COUNT = head_size / QK_VEC_SIZE;
  static_assert(QK_THREAD_COUNT <= THREADS_PER_BLOCK);

  // The layout of the cache is [B, H, head_size/x, L, x] with x == 4/8/16 for FP32/FP16/FP8. Since each thread
  // owns x elements, we have to decompose the linear index into chunks of x values and the posi-
  // tion of the thread in that chunk.

  static_assert(sizeof(T) <= 16);
  static_assert(sizeof(Qk_vec_m) <= 16);

  // The number of elements in a chunk of 16B (that's the x in the above formula).
  constexpr int QK_ELTS_IN_16B = 16 / sizeof(T);

  // The number of K vectors in 16B.
  constexpr int QK_VECS_IN_16B = 16 / sizeof(Qk_vec_m);

  // The batch/beam idx
  const int bi = blockIdx.y;

  // The beam idx
  // const int beami = bi % params.beam_width;

  // The "beam-aware" batch idx
  const int bbi = bi / params.beam_width;

  // The head.
  const int hi = blockIdx.x;

  // Combine the batch and the head indices.
  const int bhi = bi * params.num_heads + hi;

  // Combine the "beam-aware" batch idx and the head indices.
  const int bbhi = bbi * params.beam_width * params.num_heads + hi;

  const int input_beam_index = bi % params.beam_width;

  // The thread in the block.
  const int tidx = threadIdx.x;

  // While doing the product Q*K^T for the different keys we track the max.
  float qk_max = -FLT_MAX;

  float qk = 0.0F;

  int qkv_base_offset = params.is_mha && !params.is_packed_qkv
                            ? bi * params.hidden_size + hi * head_size
                            : bi * (3 * params.hidden_size) + hi * head_size;

  const size_t bi_total_seq_length = bi * params.total_sequence_length;

  const size_t bi_max_seq_length = bi * params.max_sequence_length;

  int tlength = params.is_cross_attention ? params.kv_sequence_length : params.past_sequence_length;

  // First QK_THREAD_COUNT load Q and K + the bias values for the current timestep.
  const bool is_active_qk_thread = tidx < QK_THREAD_COUNT;

  // The offset in the Q and K buffer also accounts for the batch.
  int qk_offset = qkv_base_offset + tidx * QK_VEC_SIZE;

  // Trigger the loads from the Q and K buffers.
  Qk_vec_k q;
  zero(q);

  if (is_active_qk_thread) {
    q = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.q)[qk_offset]));
  }

  // The offset in the bias buffer.
  int qk_bias_offset = hi * head_size + tidx * QK_VEC_SIZE;

  // Trigger the loads from the Q and K bias buffers.
  if (params.q_bias && is_active_qk_thread) {
    Qk_vec_k q_bias;

    q_bias = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.q_bias)[qk_bias_offset]));

    q = add_vec(q, q_bias);
  }

  TQ8* params_k_cache = reinterpret_cast<TQ8*>(params.k_cache);

  const float inv_sqrt_dh = params.scale;

  if (is_active_qk_thread) {
    // Store the Q values to shared memory.
    *reinterpret_cast<Qk_vec_k*>(&q_smem[tidx * QK_VEC_SIZE]) = q;
  }

  if (!params.is_cross_attention) {
    Qk_vec_k k;

    zero(k);

    if (is_active_qk_thread) {
      k = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.k)[qk_offset]));

      if (params.k_bias) {
        Qk_vec_k k_bias;

        k_bias = vec_conversion<Qk_vec_k, Qk_vec_m>(*reinterpret_cast<const Qk_vec_m*>(&reinterpret_cast<T*>(params.k_bias)[qk_bias_offset]));

        k = add_vec(k, k_bias);
      }
    }

    if (params.rotary_embedding_dim > 0) {
      const bool do_rotary = is_active_qk_thread && QK_VEC_SIZE * tidx < params.rotary_embedding_dim;

      T* q_smem = reinterpret_cast<T*>(smem_);
      T* k_smem = q_smem + params.rotary_embedding_dim;

      const int half_rotary_dim = params.rotary_embedding_dim / 2;
      const int half_idx = (tidx * QK_VEC_SIZE) / half_rotary_dim;
      const int intra_half_idx = (tidx * QK_VEC_SIZE) % half_rotary_dim;
      const int smem_pitch = half_rotary_dim;

      assert(half_rotary_dim % QK_VEC_SIZE == 0);

      if (do_rotary) {
        *reinterpret_cast<Qk_vec_k*>(q_smem + half_idx * smem_pitch + intra_half_idx) = q;
        *reinterpret_cast<Qk_vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx) = k;
      }

      __syncthreads();

      const int transpose_idx = half_idx * (half_rotary_dim / 2) + intra_half_idx / 2;
      constexpr int tidx_factor = (QK_VEC_SIZE > 1) ? QK_VEC_SIZE / 2 : 1;

      if (do_rotary) {
        vec_from_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
        vec_from_smem_transpose(k, k_smem, transpose_idx, smem_pitch);

        apply_rotary_embedding(
            q, k, transpose_idx / tidx_factor, params.rotary_embedding_dim, params.t_step);

        write_smem_transpose(k, k_smem, transpose_idx, smem_pitch);
        write_smem_transpose(q, q_smem, transpose_idx, smem_pitch);
      }

      __syncthreads();

      if (do_rotary) {
        q = *reinterpret_cast<Qk_vec_k*>(q_smem + half_idx * smem_pitch + intra_half_idx);
        k = *reinterpret_cast<Qk_vec_k*>(k_smem + half_idx * smem_pitch + intra_half_idx);
      }

      __syncthreads();
    }

    TFp max_abs_k = MaxAbsFloat(k);
    // Perform the final reduction to compute the max inside each warp.
    const int qk_threads_per_scale = params.quant_kv_block_size / QK_VEC_SIZE;
    if (qk_threads_per_scale <= WARP_SIZE) {
      for (int mask = qk_threads_per_scale / 2; mask >= 1; mask /= 2) {
        max_abs_k = __hmax(max_abs_k, __shfl_xor_sync(uint32_t(-1), max_abs_k, mask, qk_threads_per_scale));
      }
    } else {
      assert(qk_threads_per_scale / WARP_SIZE == 2);
      float max_abs_k_fp32 = (float)max_abs_k;
      max_abs_k = (TFp)block_sum<2>(&red_smem[2], max_abs_k_fp32);
    }

    if (is_active_qk_thread) {
      // Write the K values to the global memory cache.
      // NOTE: The stores are uncoalesced as we have multiple chunks of 16B spread across the memory
      // system. We designed it this way as it allows much better memory loads (and there are many
      // more loads) + the stores are really "write and forget" since we won't need the ack before
      // the end of the kernel. There's plenty of time for the transactions to complete.

      // The 16B chunk written by the thread.
      int co = tidx / QK_VECS_IN_16B;

      // The position of the thread in that 16B chunk.
      int ci = tidx % QK_VECS_IN_16B * QK_VEC_SIZE;

      // Two chunks are separated by L * x elements. A thread write QK_VEC_SIZE elements.
      int offset = bhi * params.max_sequence_length * head_size + co * params.max_sequence_length * QK_ELTS_IN_16B +
                   tlength * QK_ELTS_IN_16B + ci;
      // Trigger the stores to global memory.
      QuantizeTo(&params_k_cache[offset], vec_conversion<Qk_vec_m, Qk_vec_k>(k), __half2half2(max_abs_k));
      if (tidx % qk_threads_per_scale == 0) {
        const int scale_offset = (bhi * params.max_sequence_length + tlength) *scales_per_head + tidx / qk_threads_per_scale;
        *(((TFp*)params.k_scale) + scale_offset) = (TFp)max_abs_k;
      }

      // Compute \sum_i Q[i] * K^T[i] for the current timestep.
      using Qk_vec_acum = Qk_vec_k;
      qk = dot<Qk_vec_acum, Qk_vec_k>(q, k);

      if (QK_THREAD_COUNT <= WARP_SIZE) {
#pragma unroll
        for (int mask = QK_THREAD_COUNT / 2; mask >= 1; mask /= 2) {
          qk += __shfl_xor_sync(shfl_mask(QK_THREAD_COUNT), qk, mask);
        }
      }
    }

    if (QK_THREAD_COUNT > WARP_SIZE) {
      constexpr int WARPS_PER_RED = (QK_THREAD_COUNT + WARP_SIZE - 1) / WARP_SIZE;
      qk = block_sum<WARPS_PER_RED>(&red_smem[WARPS_PER_RED], qk);
    }

    // Store that value in shared memory. Keep the Q*K^T value in register for softmax.
    if (tidx == 0) {
      // Normalize qk.
      qk *= inv_sqrt_dh;
      if (params.relative_attention_bias != nullptr) {
        qk = add_vec(qk,
                     reinterpret_cast<T*>(params.relative_attention_bias)[hi * params.sequence_length * params.total_sequence_length + tlength]);
      }
      qk_max = qk;
      qk_smem[tlength] = qk;
    }
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The type of queries and keys for the math in the Q*K^T product.
  using K_vec_k = typename K_vec_k_<T, THREADS_PER_KEY>::Type;
  using K_vec_m = typename K_vec_m_<T, THREADS_PER_KEY>::Type;

  // The number of elements per vector.
  constexpr int K_VEC_SIZE = sizeof(K_vec_m) / sizeof(T);

  // Make sure the hidden size per head is a multiple of the vector size.
  static_assert(head_size % K_VEC_SIZE == 0, "");

  // The number of elements per thread.
  constexpr int K_ELTS_PER_THREAD = head_size / THREADS_PER_KEY;

  // The number of vectors per thread.
  constexpr int K_VECS_PER_THREAD = K_ELTS_PER_THREAD / K_VEC_SIZE;

  // The position the first key loaded by each thread from the cache buffer (for this B * H).
  int ko = tidx / THREADS_PER_KEY;

  // The position of the thread in the chunk of keys.
  int ki = tidx % THREADS_PER_KEY * K_VEC_SIZE;

  static_assert(head_size == THREADS_PER_KEY * K_VEC_SIZE * K_VECS_PER_THREAD);

  // Load the Q values from shared memory. The values are reused during the loop on K.
  K_vec_k q_vec[K_VECS_PER_THREAD];
#pragma unroll
  for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
    q_vec[ii] = *reinterpret_cast<const K_vec_k*>(&q_smem[ki + ii * THREADS_PER_KEY * K_VEC_SIZE]);
  }

  // The number of timesteps loaded per iteration.
  constexpr int K_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_KEY;

  // The number of keys per warp.
  constexpr int K_PER_WARP = WARP_SIZE / THREADS_PER_KEY;

  // Base pointer for the beam's batch, before offsetting with indirection buffer
  TQ8* k_cache_batch = &params_k_cache[bbhi * params.max_sequence_length * head_size + ki];

  // Pick a number of keys to make sure all the threads of a warp enter (due to shfl_sync).
  int ti_end = ((tlength + K_PER_WARP - 1) / K_PER_WARP) * K_PER_WARP;

  // Iterate over the keys/timesteps to compute the various (Q*K^T)_{ti} values.
  bool has_beams = params.cache_indir != nullptr && !params.is_cross_attention;
  const int* beam_indices = has_beams ? &params.cache_indir[bi_max_seq_length] : nullptr;

  for (int ti = ko; ti < ti_end; ti += K_PER_ITER) {
    bool is_masked = (params.mask != nullptr) && (params.mask[bi_total_seq_length + ti] == 0);
    const int mapped_beam_index = (has_beams && ti < tlength) ? beam_indices[ti] : input_beam_index;
    const int beam_offset = mapped_beam_index * params.num_heads * params.max_sequence_length * head_size;

    const int mapped_bhi = bbhi + mapped_beam_index * params.num_heads;
    int scale_offset = (mapped_bhi * params.max_sequence_length + ti) * scales_per_head + ki / params.quant_kv_block_size;
    TFp scale_of_k = ((ti < tlength) ? *(((TFp*)params.k_scale) + scale_offset) : TFp{0.0});

    // The keys loaded from the key cache.
    float qk = 0.0;

    if (ti < tlength) {
      using QuantK_Vec_m = typename QuantVec<K_vec_m>::Type;
      QuantK_Vec_m quant_k_vec[K_VECS_PER_THREAD];

      #pragma unroll
      for (int ii = 0; ii < K_VECS_PER_THREAD; ++ii) {
        int jj = ii * params.max_sequence_length + ti;
        quant_k_vec[ii] = *(const QuantK_Vec_m *)&k_cache_batch[beam_offset + jj * QK_ELTS_IN_16B];
      }

      using K_vec_acum = K_vec_k;
      K_vec_k k_vec = vec_conversion<K_vec_k, K_vec_m>(DequantizeVec<K_vec_m>(quant_k_vec[0], __half2half2(scale_of_k)));
      K_vec_acum qk_acc = mul<K_vec_acum, K_vec_k, K_vec_k>(q_vec[0], k_vec);

      #pragma unroll
      for (int ii = 1; ii < K_VECS_PER_THREAD; ++ii) {
        k_vec = vec_conversion<K_vec_k, K_vec_m>(DequantizeVec<K_vec_m>(quant_k_vec[ii], __half2half2(scale_of_k)));
        qk_acc = onnxruntime::cuda::fma(q_vec[ii], k_vec, qk_acc);
      }

      qk = sum(qk_acc);
    }

    #pragma unroll
    for (int mask = THREADS_PER_KEY / 2; mask >= 1; mask /= 2) {
      qk += __shfl_xor_sync(uint32_t(-1), qk, mask);
    }
    qk *= inv_sqrt_dh;

    // This is a deviation from FasterTransformer kernel implementation
    // but this aligns with ORT's other Attention kernels which strives to
    // mimic PyTorch when dealing with mask filter values
    if (is_masked) {
      qk += params.mask_filter_value;
    }

    // Store the product to shared memory. There's one qk value per timestep. Update the max.
    if (ti < tlength && tidx % THREADS_PER_KEY == 0) {
      if (params.relative_attention_bias != nullptr) {
        qk = add_vec(qk,
                     reinterpret_cast<T*>(params.relative_attention_bias)[hi * params.sequence_length * params.total_sequence_length + ti]);
      }
      qk_max = fmaxf(qk_max, qk);
      qk_smem[ti] = qk;
    }
  }

  // Perform the final reduction to compute the max inside each warp.
  //
  // NOTE: In a group of THREADS_PER_KEY threads, the leader already has the max value for the
  // group so it's not needed to run the reduction inside the group (again).
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREADS_PER_KEY; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Decompose the thread index into warp and lane.
  const int warp = tidx / WARP_SIZE;
  const int lane = tidx % WARP_SIZE;

  // The warp leader writes the max to shared memory.
  if (lane == 0) {
    red_smem[warp] = qk_max;
  }

  // Make sure the products are in shared memory.
  __syncthreads();

  // The warps finalize the reduction.
  qk_max = lane < WARPS_PER_BLOCK ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = WARPS_PER_BLOCK / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, __shfl_xor_sync(uint32_t(-1), qk_max, mask));
  }

  // Broadcast to all the threads in the warp.
  qk_max = __shfl_sync(uint32_t(-1), qk_max, 0);

  // Compute the logits and start the sum.
  float sum = 0.f;
  int sum_tlength = params.is_cross_attention ? tlength - 1 : tlength;
  for (int ti = tidx; ti <= sum_tlength; ti += THREADS_PER_BLOCK) {
    // This is a deviation from FasterTransformer kernel implementation
    // but this aligns with ORT's other Attention kernels which strives to
    // mimic PyTorch when dealing with mask filter values
    float logit = __expf(qk_smem[ti] - qk_max);
    sum += logit;
    qk_smem[ti] = logit;
  }

  // Compute the sum.
  sum = block_sum<WARPS_PER_BLOCK>(&red_smem[WARPS_PER_BLOCK], sum);

  // Normalize the logits.
  float inv_sum = __fdividef(1.f, sum + 1.e-6f);
  for (int ti = tidx; ti <= sum_tlength; ti += THREADS_PER_BLOCK) {
    float logit = qk_smem[ti] * inv_sum;
    ConvertFromFloat(logits_smem[ti], logit);
  }

  // Put Values part below so we leverage __syncthreads
  // from the previous step

  // The number of elements per vector.
  constexpr int V_VEC_SIZE = head_size / THREADS_PER_VALUE;

  // A vector of V elements for the current timestep.
  using V_vec_k = typename V_vec_k_<T, V_VEC_SIZE>::Type;
  using V_vec_m = typename V_vec_m_<T, V_VEC_SIZE>::Type;

  // The value computed by this thread.
  int vo = tidx / THREADS_PER_VALUE;

  // The hidden dimensions computed by this particular thread.
  int vi = tidx % THREADS_PER_VALUE * V_VEC_SIZE;

  // The base pointer for the value in the cache buffer.
  TQ8* params_v_cache = reinterpret_cast<TQ8*>(params.v_cache);

  TQ8* v_cache = &params_v_cache[bhi * params.max_sequence_length * head_size + vi];

  // Base pointer for the beam's batch, before offsetting with indirection buffer
  TQ8* v_cache_batch = &params_v_cache[bbhi * params.max_sequence_length * head_size + vi];

  // The number of values processed per iteration of the loop.
  constexpr int V_PER_ITER = THREADS_PER_BLOCK / THREADS_PER_VALUE;

  // One group of threads computes the product(s) for the current timestep.
  V_vec_k v_bias;
  if (params.v_bias && !params.is_cross_attention) {
    zero(v_bias);

    T* params_v_bias = reinterpret_cast<T*>(params.v_bias);

    if (vo == tlength % V_PER_ITER) {
      v_bias = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&params_v_bias[hi * head_size + vi]));
    }
  }

  // From previous, before values, step
  // Also make sure the logits are in shared memory.
  __syncthreads();

  using V_vec_acum = typename V_vec_acum_fp32_<V_vec_k>::Type;

  // The partial outputs computed by each thread.
  V_vec_acum out;
  zero(out);

  // Loop over the timesteps to compute the partial outputs.
  for (int ti = vo; ti < tlength; ti += V_PER_ITER) {
    // Fetch offset based on cache_indir when beam sampling
    const int mapped_beam_index = has_beams ? params.cache_indir[bi_max_seq_length + ti] : input_beam_index;
    const int beam_offset = mapped_beam_index * params.num_heads * params.max_sequence_length * head_size;

    const int mapped_bhi = bbhi + mapped_beam_index * params.num_heads;
    const int scale_offset = (mapped_bhi * params.max_sequence_length + ti) * scales_per_head + vi / params.quant_kv_block_size;
    TFp scale_of_v = *(((TFp*)params.v_scale) + scale_offset);

    // Load the values from the cache.
    // V_vec_k v = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&v_cache_batch[beam_offset + ti * head_size]));
    V_vec_k v = vec_conversion<V_vec_k, V_vec_m>(LoadQ8(
                    reinterpret_cast<const V_vec_m*>(&v_cache_batch[beam_offset + ti * head_size]), __half2half2(scale_of_v)));

    // Load the logits from shared memory.
    T logit = logits_smem[ti];
    out = fma(logit, v, out);
  }

  // One group of threads computes the product(s) for the current timestep.
  if (vo == tlength % V_PER_ITER && !params.is_cross_attention) {
    const auto v_offset = qkv_base_offset + vi;

    V_vec_k v;
    v = vec_conversion<V_vec_k, V_vec_m>(*reinterpret_cast<const V_vec_m*>(&reinterpret_cast<T*>(params.v)[v_offset]));
    if (params.v_bias) {
      v = add_vec(v, v_bias);
    }

    static_assert(THREADS_PER_VALUE <= WARP_SIZE / 2);
    TFp max_abs_v = MaxAbsFloat(v);
    const uint32_t group_id = (tidx % WARP_SIZE) / THREADS_PER_VALUE;
    const uint32_t group_masks =  ((1u << THREADS_PER_VALUE) - 1) << (group_id * THREADS_PER_VALUE);
    #pragma unroll
    for (int mask = THREADS_PER_VALUE / 2; mask >= 1; mask /= 2) {
      max_abs_v = __hmax(max_abs_v, __shfl_xor_sync(group_masks, max_abs_v, mask, THREADS_PER_VALUE));
    }

    // Store the values with bias back to global memory in the cache for V.
    //*reinterpret_cast<V_vec_m*>(&v_cache[tlength * head_size]) = vec_conversion<V_vec_m, V_vec_k>(v);
    QuantizeTo(&v_cache[tlength * head_size], vec_conversion<V_vec_m, V_vec_k>(v), __half2half2(max_abs_v));
    if (vi % params.quant_kv_block_size == 0) {
      const int scales_per_head = head_size / params.quant_kv_block_size;
      const int scale_offset = (bhi * params.max_sequence_length + tlength) * scales_per_head + vi / params.quant_kv_block_size;
      *(((TFp*)params.v_scale) + scale_offset) = (TFp)max_abs_v;
    }

    // Initialize the output value with the current timestep.
    out = fma(logits_smem[tlength], v, out);
  }

  // Make sure we can start writing to shared memory.
  __syncthreads();

  // Run the final reduction amongst the different groups computing different partial outputs.
#pragma unroll
  for (int active_groups = V_PER_ITER; active_groups >= 2; active_groups /= 2) {
    // The midpoint in the number of active groups.
    int midpoint = active_groups / 2;

    // The upper part of active threads store to shared memory.
    if (vo >= midpoint && vo < active_groups) {
      ConvertFromFloat(*reinterpret_cast<V_vec_k*>(&out_smem[(vo - midpoint) * head_size + vi]), out);
    }
    __syncthreads();

    // The bottom warps update their values.
    if (vo < midpoint) {
      out = add_vec(*reinterpret_cast<const V_vec_k*>(&out_smem[vo * head_size + vi]), out);
    }
    __syncthreads();
  }

  // Output the final values.
  T* params_out = reinterpret_cast<T*>(params.out);
  if (vo == 0) {
    ConvertFromFloat(*reinterpret_cast<V_vec_m*>(&params_out[bhi * head_size + vi]), out);
  }
#endif
}

// Template instantiation(s)

// fp32 + head size = 32
// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 32, 4, 8, 64>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 32, 2, 8, 128>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 32, 1, 8, 256>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// fp16 + head size = 32
template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 32, 4, 4, 64>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 32, 2, 4, 128>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 32, 1, 4, 256>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// fp32 + head size = 64
// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 64, 4, 16, 64>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 64, 2, 16, 128>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 64, 1, 16, 256>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// fp16 + head size = 64
template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 64, 4, 8, 64>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 64, 2, 8, 128>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 64, 1, 8, 256>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// fp32 + head size = 128
// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 128, 4, 32, 64>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 128, 2, 32, 128>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// template void __global__ masked_multihead_attention_quant_kv_kernel<float, 128, 1, 32, 256>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

// fp16 + head size = 128
template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 128, 4, 16, 64>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 128, 2, 16, 128>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

template void __global__ masked_multihead_attention_quant_kv_kernel<uint16_t, 128, 1, 16, 256>(DecoderMaskedMultiHeadAttentionQuantKVParams params);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
