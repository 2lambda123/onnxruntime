#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on qkvToContext plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/

Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Modifications:
// (1) support GPT-2 past state, unidirectional mask and 4D attention mask from Megatron
// (2) support 2D attention mask
// (3) allow persistent softmax from PyTorch for debugging purpose.
// (4) support different input hidden size and model hidden size for pruned model
// (5) support different hidden sizes of Q/K and V
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <cassert>
#include <hip/hip_fp16.h>
#include <hipcub/hipcub.hpp>
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/shared_inc/fpgeneric.h"
#include "contrib_ops/cuda/bert/attention_softmax.h"
#include "contrib_ops/cuda/bert/transformer_common.h"
#include "contrib_ops/cuda/bert/add_bias_transpose.h"
// #include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/mha_runner.h"
// #include "contrib_ops/cuda/bert/tensorrt_fused_multihead_attention/cross_attention/fmha_cross_attention.h"
#include "contrib_ops/cpu/bert/attention_base.h"
#include "contrib_ops/cuda/bert/bert_padding.h"
#include "contrib_ops/cuda/transformers/dump_cuda_tensor.h"
// #include "contrib_ops/cuda/bert/cutlass_fmha/memory_efficient_attention.h"
#include "contrib_ops/cuda/bert/flash_attention/flash_api.h"
#include "contrib_ops/cuda/bert/group_query_attention_impl.h"
#include "contrib_ops/cuda/bert/attention_impl.h"

using namespace onnxruntime::cuda;
using namespace onnxruntime::contrib::attention_softmax_cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

// Kernel for seqlens_k
__global__ void repeat_seqlen(int32_t* seqlens_k, int32_t seqlen, int batch_size) {
  int id = blockDim.x * blockIdx.x + threadIdx.x;
  if(id < batch_size) seqlens_k[id] = seqlen;
}

template <typename T>
Status QkvToContext(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<T>& data) {
  auto stream = static_cast<hipStream_t>(ort_stream->GetHandle());
  const int batch_size = parameters.batch_size;
  const int sequence_length = parameters.sequence_length;
  const int kv_sequence_length = parameters.kv_sequence_length;
  const int max_sequence_length = parameters.max_sequence_length;
  const int num_heads = parameters.num_heads;
  const int kv_num_heads = parameters.kv_num_heads;
  const int head_size = parameters.head_size;

  AttentionQkvFormat qkv_format = AttentionQkvFormat::Q_K_V_BSNH;

  // For raw attention mask, the scalar 1/sqrt(H) is moved to combine with softmax computation.
  const float scale = parameters.scale == 0.0f ? 1.f / sqrt(static_cast<float>(head_size)) : parameters.scale;
  assert(data.use_flash_attention);
#if USE_FLASH_ATTENTION
  if (data.use_flash_attention) {
    assert(qkv_format == AttentionQkvFormat::Q_K_V_BSNH);
    assert(parameters.num_heads % parameters.kv_num_heads == 0);

    void* query = reinterpret_cast<void*>(const_cast<T*>(data.query));
    void* key = reinterpret_cast<void*>(const_cast<T*>(data.key));
    void* value = reinterpret_cast<void*>(const_cast<T*>(data.value));

    bool is_causal = parameters.is_unidirectional;

    if (data.past_key == nullptr) {
      // TODO(aciddelgado): add support for concatenating past and kv to present kv when seqlens_k is not given
      ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd(
          device_prop, stream, query, key, value, data.output, reinterpret_cast<void*>(data.softmax_lse),
          parameters.batch_size, parameters.num_heads, parameters.kv_num_heads, head_size,
          parameters.sequence_length, parameters.total_sequence_length, scale, is_causal, parameters.num_splits,
          reinterpret_cast<void*>(data.softmax_lse_accum), reinterpret_cast<void*>(data.out_accum)));
    } else {
      // Assume past and present kv share buffer.
      assert(parameters.past_sequence_length >= 0);
      assert(data.past_value != nullptr);

      void* past_key = reinterpret_cast<void*>(const_cast<T*>(data.past_key));
      void* past_value = reinterpret_cast<void*>(const_cast<T*>(data.past_value));

      // Launch kernel to copy seqlen
      int thr_per_blk = 256;
      int blk_in_grid = ceil( float(batch_size) / thr_per_blk );
      repeat_seqlen<<< blk_in_grid, thr_per_blk, 0, stream >>>(data.seqlens_k, parameters.past_sequence_length, batch_size);

      ORT_RETURN_IF_ERROR(onnxruntime::flash::mha_fwd_kvcache(
          device_prop, stream, query, past_key, past_value, key, value, data.output, reinterpret_cast<void*>(data.softmax_lse),
          reinterpret_cast<void*>(data.seqlens_k), batch_size, num_heads, kv_num_heads,
          head_size, sequence_length, max_sequence_length, kv_sequence_length,
          scale, is_causal, parameters.num_splits, reinterpret_cast<void*>(data.softmax_lse_accum),
          reinterpret_cast<void*>(data.out_accum)));
    }


    DUMP_TENSOR("flash attention output", data.output, batch_size, sequence_length, num_heads, head_size);

    return Status::OK();
  }
#endif
  return ORT_MAKE_STATUS(ONNXRUNTIME, INVALID_ARGUMENT, "Unfused Group Query Attention not implemented yet.");

  // // The following are unfused attention.
  // assert(qkv_format == AttentionQkvFormat::Q_K_V_BNSH);

  // // Raw attention mask could be 2D (BxT) or 3D (BxSxT) or 4D(Bx1xMxM), where M is the max sequence length.
  // bool use_raw_attention_mask = (nullptr != mask_index && mask_index_dims.size() >= 2);

  // // Compute Q*K' (as K'*Q), scaled by 1/sqrt(H) and store in scratch1: BxNxSxT
  // // Q: BxNxSxH, K (present_k): BxNxTxH, Q*K': BxNxSxT
  // float one = 1.0f;
  // float zero = 0.f;

  // float alpha = use_raw_attention_mask ? one : scale;

  // hipblasSetStream(cublas, stream);

  // DUMP_TENSOR_D("q[BNSH]", q, batch_size, num_heads, sequence_length, qk_head_size);
  // DUMP_TENSOR_D("k[BNSH]", k, batch_size, num_heads, total_sequence_length, qk_head_size);
  // CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
  //     cublas, HIPBLAS_OP_T, HIPBLAS_OP_N,
  //     total_sequence_length, sequence_length, qk_head_size,
  //     &alpha, k, qk_head_size, present_size_per_batch_k,
  //     q, qk_head_size, sequence_length * qk_head_size,
  //     &zero, scratch1, total_sequence_length, sequence_length * total_sequence_length, batches, device_prop));

  // DUMP_TENSOR_D("Q", q, batch_size, num_heads, sequence_length, qk_head_size);
  // DUMP_TENSOR_D("K", k, batch_size, num_heads, qk_head_size, sequence_length);
  // DUMP_TENSOR_D("QK", scratch1, batch_size, num_heads, sequence_length, total_sequence_length);

  // const size_t bytes = GetAttentionScratchSize(element_size, batch_size, num_heads,
  //                                              sequence_length, total_sequence_length);
  // T* scratch2 = scratch1 + (bytes / element_size);

  // // Apply softmax and store result R to scratch2: BxNxSxT
  // if (use_raw_attention_mask) {  // 2d, 3d or 4d attention mask
  //   const int mask_dimension = static_cast<int>(mask_index_dims.size());

  //   // For testing, environment variable ORT_TRANSFORMER_OPTIONS=1 could enable persistent softmax used in Torch.
  //   const TransformerOptions* options = TransformerOptions::GetInstance();
  //   bool use_persistent_softmax = options->IsPrecisionMode() && !options->DisablePersistentSoftmax();

  //   T* persistent_softmax_workspace = scratch1;  // replace Q*K' in place with masked score for persistent softmax.
  //   ORT_RETURN_IF_ERROR(
  //       ComputeSoftmaxWithRawMask<T>(
  //           ort_stream, total_sequence_length, sequence_length, batch_size, num_heads,
  //           mask_index, nullptr, data.relative_position_bias, parameters.broadcast_res_pos_bias,
  //           scratch1, scratch2, parameters.is_unidirectional, scale, mask_dimension,
  //           parameters.max_sequence_length, use_persistent_softmax, persistent_softmax_workspace,
  //           mask_filter_value));
  // } else if (nullptr != mask_index) {  // 1d mask index
  //   assert(mask_index_dims.size() == 1);
  //   // mask_index has 1D shape: either (batch_size) or (2*batch_size). Only the later one has start postions.
  //   const int* mask_start = (mask_index_dims[0] > batch_size) ? mask_index + batch_size : nullptr;
  //   ORT_RETURN_IF_ERROR(ComputeSoftmaxWithMask1D<T>(
  //       stream, total_sequence_length, sequence_length, batch_size, num_heads,
  //       mask_index, mask_start, data.relative_position_bias, parameters.broadcast_res_pos_bias,
  //       scratch1, scratch2, parameters.is_unidirectional));
  // } else {  // no mask
  //   ORT_RETURN_IF_ERROR(
  //       ComputeSoftmax<T>(
  //           stream, total_sequence_length, sequence_length, batch_size, num_heads, data.relative_position_bias,
  //           parameters.broadcast_res_pos_bias, scratch1, scratch2, parameters.is_unidirectional));
  // }

  // DUMP_TENSOR_D("Softmax", scratch2, batch_size, num_heads, sequence_length, total_sequence_length);
  // DUMP_TENSOR_D("V", v, batch_size, num_heads, sequence_length, v_head_size);

  // // compute R*V (as V*R), and store in temp_output (space used by Q): BxNxSxH_v
  // T* temp_output = qkv;
  // CUBLAS_RETURN_IF_ERROR(cublasGemmStridedBatchedHelper(
  //     cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
  //     v_head_size, sequence_length, total_sequence_length,
  //     &one, v, v_head_size, present_size_per_batch_v,
  //     scratch2, total_sequence_length, sequence_length * total_sequence_length,
  //     &zero, temp_output, v_head_size, sequence_length * v_head_size, batches, device_prop));

  // // Temp_output is BxNxSxH_v, transpose to output BxSxNxH_v
  // Status result = LaunchTransCtx(stream, sequence_length, batch_size, v_head_size, num_heads,
  //                                max_threads_per_block, false, temp_output, data.output);
  // DUMP_TENSOR("unfused output", data.output, batch_size, sequence_length, num_heads, v_head_size);
  // return result;
}

// Template Instantiation
// template struct AttentionData<float>;

template struct GroupQueryAttentionData<half>;

// template Status QkvToContext<float>(
//     const hipDeviceProp_t& device_prop,
//     hipblasHandle_t& cublas,
//     Stream* ort_stream,
//     contrib::AttentionParameters& parameters,
//     AttentionData<float>& data);

template Status QkvToContext<half>(
    const hipDeviceProp_t& device_prop,
    hipblasHandle_t& cublas,
    Stream* ort_stream,
    contrib::GroupQueryAttentionParameters& parameters,
    GroupQueryAttentionData<half>& data);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
